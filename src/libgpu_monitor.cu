#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <random>
#include <string>
#include <ctime>
#include <chrono>
#include <thread>
#include <gpu_monitor/gpu_monitor.hpp>

#ifdef ACC_CUDA
#include "gpu_monitor_cuda.hpp"
#endif

#ifdef ACC_HIP
#include "gpu_monitor_hip.hpp"
#endif

std::vector<mtk::gpu_monitor::profiling_data> mtk::gpu_monitor::measure_power_consumption(
		const std::function<void(void)> func,
		const std::time_t interval
		) {
	std::vector<mtk::gpu_monitor::profiling_data> profiling_result;

#ifdef ACC_CUDA
	mtk::gpu_monitor::gpu_monitor_cuda gpu_monitor;
#endif
#ifdef ACC_HIP
	mtk::gpu_monitor::gpu_monitor_hip gpu_monitor;
#endif
	gpu_monitor.init();
	int gpu_id = gpu_monitor.get_current_device();
	// Output log
	unsigned count = 0;

	int semaphore = 1;

	// Start target thread
	std::thread thread(
			[&](){func();semaphore = 0;}
			);

	// Start measurement
	const auto start_clock = std::chrono::high_resolution_clock::now();
	do {
		const auto end_clock = std::chrono::high_resolution_clock::now();
		const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count();

		// call measurement functions
		const auto temperature = gpu_monitor.get_current_temperature(gpu_id);
		const auto power = gpu_monitor.get_current_power(gpu_id);
		const auto memory_consumption = gpu_monitor.get_current_used_memory(gpu_id);

		const auto end_clock_1 = std::chrono::high_resolution_clock::now();
		const auto elapsed_time_1 = std::chrono::duration_cast<std::chrono::milliseconds>(end_clock_1 - start_clock).count();

		// Store data
		profiling_result.push_back(mtk::gpu_monitor::profiling_data{temperature, power, memory_consumption, static_cast<std::time_t>(elapsed_time)});

		// Sleep
		using namespace std::chrono_literals;
		std::this_thread::sleep_for(std::chrono::milliseconds(std::max<std::time_t>(static_cast<int>(interval) * count, elapsed_time_1) - elapsed_time_1));
		count++;
	} while (semaphore);

	thread.join();

	gpu_monitor.shutdown();

	return profiling_result;
}

double mtk::gpu_monitor::get_integrated_power_consumption(
		const std::vector<mtk::gpu_monitor::profiling_data>& profiling_data_list
		) {
	if (profiling_data_list.size() == 0) {
		return 0.0;
	}

	double power_consumption = 0.;
	for (unsigned i = 1; i < profiling_data_list.size(); i++) {
		const auto elapsed_time = (profiling_data_list[i].timestamp - profiling_data_list[i - 1].timestamp) * 1e-6;
		// trapeziodal integration
		power_consumption += (profiling_data_list[i].power + profiling_data_list[i - 1].power) / 2 * elapsed_time;
	}
	return power_consumption;
}

double mtk::gpu_monitor::get_elapsed_time(
		const std::vector<mtk::gpu_monitor::profiling_data>& profiling_data_list
		) {
	if (profiling_data_list.size() == 0) {
		return 0.0;
	}
	return (profiling_data_list[profiling_data_list.size() - 1].timestamp - profiling_data_list[0].timestamp) * 1e-6;
}
