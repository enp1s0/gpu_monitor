#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <random>
#include <string>
#include <ctime>
#include <chrono>
#include <thread>
#include <gpu_monitor/gpu_monitor.h>
#include <unistd.h>

#ifdef ACC_CUDA
#include "gpu_monitor_cuda.hpp"
#endif

#ifdef ACC_HIP
#include "gpu_monitor_hip.hpp"
#endif

std::vector<std::tuple<std::time_t, double, double, double>> mtk::gpu_monitor::measure_power_consumption(
		const std::function<void(void)> func,
		const std::time_t interval
		) {
	std::vector<std::tuple<std::time_t, double, double, double>> profiling_result;

#ifdef ACC_CUDA
	mtk::gpu_monitor::gpu_monitor_cuda gpu_monitor;
#endif
#ifdef ACC_HIP
	mtk::gpu_monitor::gpu_monitor_hip gpu_monitor;
#endif
	gpu_monitor.init();
	int gpu_id = gpu_monitor.get_current_device();
	// Output log
	unsigned count = 0;

	// Start target thread
	std::thread thread(func);

	// Start measurement
	const auto start_clock = std::chrono::high_resolution_clock::now();
	do {
		const auto end_clock = std::chrono::high_resolution_clock::now();
		const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count();

		// call measurement functions
		const auto temperature = gpu_monitor.get_current_temperature(gpu_id);
		const auto power = gpu_monitor.get_current_power(gpu_id);
		const auto memory_consumption = gpu_monitor.get_current_used_memory(gpu_id);

		const auto end_clock_1 = std::chrono::high_resolution_clock::now();
		const auto elapsed_time_1 = std::chrono::duration_cast<std::chrono::microseconds>(end_clock_1 - start_clock).count();

		// Store data
		profiling_result.push_back(std::make_tuple(temperature, power, memory_consumption, static_cast<std::time_t>(elapsed_time_1)));

		// Sleep
		usleep(std::max<std::time_t>(interval * 1000 * count, elapsed_time_1) - elapsed_time_1);
	} while (!thread.joinable());

	thread.join();

	gpu_monitor.shutdown();

	return profiling_result;
}
