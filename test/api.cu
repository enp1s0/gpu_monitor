#include "hip/hip_runtime.h"
#include <iostream>
#include <gpu_monitor/gpu_monitor.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>

constexpr std::size_t N = 1lu << 15;

int main() {
	auto mat = cutf::memory::get_device_unique_ptr<float>(N * N);

	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();

	auto lauch_gemm = [&]() {
		const float a = 1.0f;
		for (unsigned i = 0; i < 10; i++) {
			CUTF_CHECK_ERROR(cutf::cublas::gemm(
						*cublas_handle.get(),
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						N, N, N,
						&a,
						mat.get(), N,
						mat.get(), N,
						&a,
						mat.get(), N
						));
		}
		hipDeviceSynchronize();
	};

	// Profiling
	const auto progiling_result = mtk::gpu_monitor::measure_power_consumption(
			lauch_gemm,
			20
			);

	// Get max temperature and power
	double max_temperature = 0.;
	double max_power = 0.;
	for (const auto& pr : progiling_result) {
		max_temperature = std::max(max_temperature, pr.temperature);
		max_power       = std::max(max_power      , pr.power      );
	}

	double elapsed_time = (progiling_result[progiling_result.size() - 1].timestamp - progiling_result[0].timestamp) * 1e-6;

	std::printf("Num data        : %lu\n", progiling_result.size());
	std::printf("Max power       : %e [W]\n", max_power);
	std::printf("Max temperature : %e [C]\n", max_temperature);
	std::printf("Elapsed tiem    : %e [s]\n", elapsed_time);
}
