#include "hip/hip_runtime.h"
#include <iostream>
#include <gpu_monitor/gpu_monitor.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>

constexpr std::size_t N = 1lu << 15;

int main() {
	auto mat = cutf::memory::get_device_unique_ptr<float>(N * N);

	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();

	auto lauch_gemm = [&]() {
		const float a = 1.0f;
		for (unsigned i = 0; i < 10; i++) {
			CUTF_CHECK_ERROR(cutf::cublas::gemm(
						*cublas_handle.get(),
						HIPBLAS_OP_N, HIPBLAS_OP_N,
						N, N, N,
						&a,
						mat.get(), N,
						mat.get(), N,
						&a,
						mat.get(), N
						));
		}
		hipDeviceSynchronize();
	};

	// Profiling
	const auto profiling_result = mtk::gpu_monitor::measure_power_consumption(
			lauch_gemm,
			20
			);

	// Get max temperature and power
	double max_temperature = 0.;
	double max_power = 0.;
	for (const auto& pr : profiling_result) {
		max_temperature = std::max(max_temperature, pr.temperature);
		max_power       = std::max(max_power      , pr.power      );
	}

	const auto elapsed_time = mtk::gpu_monitor::get_elapsed_time(profiling_result);
	const auto integrated_power_consumption = mtk::gpu_monitor::get_integrated_power_consumption(profiling_result);

	std::printf("Num data        : %lu\n", profiling_result.size());
	std::printf("Max power       : %e [W]\n", max_power);
	std::printf("Int power       : %e [J]\n", integrated_power_consumption);
	std::printf("Avg power       : %e [W]\n", integrated_power_consumption / elapsed_time);
	std::printf("Max temperature : %e [C]\n", max_temperature);
	std::printf("Elapsed item    : %e [s]\n", elapsed_time);
}
